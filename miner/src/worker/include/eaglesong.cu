#include "hip/hip_runtime.h"
/// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stddef.h>
#include "portable_endian.h"
#include "eaglesong.h"

#define INPUT_LEN (32)
#define N 16
#define M (INPUT_LEN >> 2)
#define OUTPUT_LEN 32
#define THREADS_PER_BLOCK  (512)
#define MAX_HASH_NUM (1<<31)
#define MAX_GPU_NUM (1024)
#define HASH_NUM (1<<29)

#define DELIMITER (0x06)

#define ROUND (43)


#define ROL32(a,b) (((a)<<(b))|((a)>>(32-(b))))
#define ROL_ADD(a,b) a += b; a = ROL32(a, 8); b = ROL32(b, 24) + a;

#define EaglesongPermutation(injection_constants) \
{ \
	for(int i = 0, j=0; i < ROUND ; ++i, j+=16) { \
		tmp = s0 ^ s4 ^ s12 ^ s15; s0 = tmp^s5 ^ s6 ^ s7; s1 = tmp^s1 ^ s8 ^ s13; \
		tmp = s1 ^ s2 ^ s6 ^ s14; s2 = tmp^s7 ^ s8 ^ s9; s3 = tmp^s3 ^ s10 ^ s15; \
		tmp = s0 ^ s3 ^ s4 ^ s8; s4 = tmp^s9 ^ s10 ^ s11; s5 = tmp^s1 ^ s5 ^ s12; \
		tmp = s2 ^ s5 ^ s6 ^ s10; s6 = tmp^s11 ^ s12 ^ s13; s7 = tmp^s3 ^ s7 ^ s14; \
		tmp = s4 ^ s7 ^ s8 ^ s12; s8 = tmp^s13 ^ s14 ^ s15; s9 = tmp^s0 ^ s5 ^ s9; \
		tmp = s6 ^ s9 ^ s10 ^ s14; s10 = tmp^s0 ^ s1 ^ s15; s11 = tmp^s2 ^ s7 ^ s11; \
		tmp = s0 ^ s8 ^ s11 ^ s12; s12 = tmp^s1 ^ s2 ^ s3; s13 = tmp^s4 ^ s9 ^ s13; \
		tmp = s3 ^ s5 ^ s13 ^ s14; s14 = tmp^s2 ^ s4 ^ s10; s15 = tmp^s0 ^ s1 ^ s6 ^ s7 ^ s8 ^ s9 ^ s15; \
		s0 ^= ROL32(s0, 2) ^ ROL32(s0, 4) ^ injection_constants[(j ^ 0)];                    \
		s1 ^= ROL32(s1, 13) ^ ROL32(s1, 22) ^ injection_constants[(j ^ 1)];                  \
		ROL_ADD(s0, s1);                                                                      \
		s2 ^= ROL32(s2, 4) ^ ROL32(s2, 19) ^ injection_constants[(j ^ 2)];                   \
		s3 ^= ROL32(s3, 3) ^ ROL32(s3, 14) ^ injection_constants[(j ^ 3)];                   \
		ROL_ADD(s2, s3);                                                                      \
		s4 ^= ROL32(s4, 27) ^ ROL32(s4, 31) ^ injection_constants[(j ^ 4)];                  \
		s5 ^= ROL32(s5, 3) ^ ROL32(s5, 8) ^ injection_constants[(j ^ 5)];                    \
		ROL_ADD(s4, s5);                                                                      \
		s6 ^= ROL32(s6, 17) ^ ROL32(s6, 26) ^ injection_constants[(j ^ 6)];                  \
		s7 ^= ROL32(s7, 3) ^ ROL32(s7, 12) ^ injection_constants[(j ^ 7)];                   \
		ROL_ADD(s6, s7);                                                                      \
		s8 ^= ROL32(s8, 18) ^ ROL32(s8, 22) ^ injection_constants[(j ^ 8)];                  \
		s9 ^= ROL32(s9, 12) ^ ROL32(s9, 18) ^ injection_constants[(j ^ 9)];                  \
		ROL_ADD(s8, s9);                                                                      \
		s10 ^= ROL32(s10, 4) ^ ROL32(s10, 7) ^ injection_constants[(j ^ 10)];                 \
		s11 ^= ROL32(s11, 4) ^ ROL32(s11, 31) ^ injection_constants[(j ^ 11)];                \
		ROL_ADD(s10, s11);                                                                    \
		s12 ^= ROL32(s12, 12) ^ ROL32(s12, 27) ^ injection_constants[(j ^ 12)];               \
		s13 ^= ROL32(s13, 7) ^ ROL32(s13, 17) ^ injection_constants[(j ^ 13)];                \
		ROL_ADD(s12, s13);                                                                    \
		s14 ^= ROL32(s14, 7) ^ ROL32(s14, 8) ^ injection_constants[(j ^ 14)];                 \
		s15 ^= ROL32(s15, 1) ^ ROL32(s15, 13) ^ injection_constants[(j ^ 15)];                \
		ROL_ADD(s14, s15); \
	} \
}

#define absorbing(s, input, i) {\
    s = (be32toh(((uint32_t*)(input))[i])); \
}

__constant__ uint32_t gpu_injection_constants[688] = INJECT_MAT;

uint32_t cpu_injection_constants[688] = INJECT_MAT;

#define squeeze(s, k) {\
    ((uint32_t *)output)[k] = (s); \
}


struct GPU_DEVICE
{
	uint32_t   state[N];
	uint32_t   nonce_id;
	uint8_t    *target;
	uint32_t    *g_state;
	uint8_t    *g_target;
	uint32_t   *g_nonce_id;
};

GPU_DEVICE *gpu_divices[MAX_GPU_NUM] = {NULL};
uint32_t gpu_divices_cnt = 0;


__global__ void g_eaglesong(uint32_t *state, uint8_t* target, uint32_t *nonce_id)
{
	uint32_t global_id = blockDim.x * blockIdx.x + threadIdx.x;

	uint32_t id = global_id % THREADS_PER_BLOCK;
	uint32_t tmp;
	uint32_t s0, s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, s11, s12, s13, s14, s15;
	uint8_t output[OUTPUT_LEN];

	__shared__ uint32_t shared_state[N];
	__shared__ uint8_t shared_target[OUTPUT_LEN];

	if (id < N) shared_state[id] = state[id]; 
	if (id < OUTPUT_LEN) shared_target[id] = target[id];
	__syncthreads();

	s0 = shared_state[0] ^ (global_id+1);
	s1 = shared_state[1]; s2 = shared_state[2]; s3 = shared_state[3];
	s4 = shared_state[4]; s5 = shared_state[5]; s6 = shared_state[6]; s7 = shared_state[7];
	s8 = shared_state[8]; s9 = shared_state[9]; s10 = shared_state[10]; s11 = shared_state[11];
	s12 = shared_state[12]; s13 = shared_state[13]; s14 = shared_state[14]; s15 = shared_state[15];
	
	EaglesongPermutation(gpu_injection_constants);

	squeeze(s0, 0); squeeze(s1, 1); squeeze(s2, 2); squeeze(s3, 3);
	squeeze(s4, 4); squeeze(s5, 5); squeeze(s6, 6); squeeze(s7, 7);

	for(int k=0; k<32; ++k) {
		if(output[k] < shared_target[k]) {
			atomicExch(nonce_id, global_id+1);
		} else if(output[k] > shared_target[k]) {
			break;
		}
	}
}

int gpu_hash(uint32_t gpuid)
{
	if ((uint32_t)HASH_NUM > (uint32_t)MAX_HASH_NUM) {
		printf("HASH_NUM out of bound!!!\n");
		return 0;
	}

	if (gpu_divices[gpuid]->g_state == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_state, sizeof(gpu_divices[gpuid]->state)) != hipSuccess) {
			printf("E01: can not find the divice specified, have you installed the right driver?\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->g_nonce_id == NULL) 
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_nonce_id, sizeof(gpu_divices[gpuid]->nonce_id)) != hipSuccess) {
			printf("E02: cuda alloc memory error for nonce\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->g_target == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_target, OUTPUT_LEN) != hipSuccess) {
			printf("E03: cuda alloc memory error for target\n");
			return 0;
		}
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_state, gpu_divices[gpuid]->state, sizeof(gpu_divices[gpuid]->state), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E04: copy memory error for state\n");
		return 0;
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_target, gpu_divices[gpuid]->target, OUTPUT_LEN, hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E05: copy memory error for target\n");
		return 0;
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_nonce_id, &(gpu_divices[gpuid]->nonce_id), sizeof(gpu_divices[gpuid]->nonce_id), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E06: copy memory error for nonce\n");
		return 0;
	}

	g_eaglesong << <HASH_NUM / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(gpu_divices[gpuid]->g_state, gpu_divices[gpuid]->g_target, gpu_divices[gpuid]->g_nonce_id);
	hipDeviceSynchronize();

	if (hipMemcpy(&(gpu_divices[gpuid]->nonce_id), gpu_divices[gpuid]->g_nonce_id, sizeof(gpu_divices[gpuid]->nonce_id), hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("E07: copy memory error for g_nonce_id\n");
		return 0;
	}

	return HASH_NUM;
}

GPU_DEVICE* New_GPU_DEVICE()
{
	GPU_DEVICE* p = NULL;
	p = (GPU_DEVICE*)malloc(sizeof(GPU_DEVICE));
	if (p != NULL)
	{
		p->g_target = NULL;
		p->g_nonce_id = NULL;
		p->g_state = NULL;
		p->g_target = NULL;
	} else {
		printf("E08: alloc memory error!\n");
	}
	return p;
}

void RESET_GPU_DEVICE(uint32_t gpuid)
{
	
	memset(gpu_divices[gpuid]->state, 0, sizeof(gpu_divices[gpuid]->state));
	gpu_divices[gpuid]->nonce_id = 0;

	hipFree(gpu_divices[gpuid]->g_nonce_id);
	hipFree(gpu_divices[gpuid]->g_state);
	hipFree(gpu_divices[gpuid]->g_target);
	
	gpu_divices[gpuid]->target = NULL;
	gpu_divices[gpuid]->g_nonce_id = NULL;
	gpu_divices[gpuid]->g_state = NULL;
	gpu_divices[gpuid]->g_target = NULL;
}

void GPU_Count()
{
	int num;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&num);
	printf("deviceCount := %d\n", num);
	gpu_divices_cnt = 0;
	for (int i = 0; i<num; i++)
	{

		hipGetDeviceProperties(&prop, i);
		printf("name:%s\n", prop.name);
		printf("totalGlobalMem:%lu GB\n", prop.totalGlobalMem / 1024 / 1024 / 1024);
		printf("multiProcessorCount:%d\n", prop.multiProcessorCount);
		printf("maxThreadsPerBlock:%d\n", prop.maxThreadsPerBlock);
		printf("sharedMemPerBlock:%lu KB\n", prop.sharedMemPerBlock/1024);
		printf("major:%d,minor:%d\n", prop.major, prop.minor);
		gpu_divices_cnt++;
	}
	if (gpu_divices_cnt > MAX_GPU_NUM)gpu_divices_cnt = MAX_GPU_NUM;
}

extern "C" {
	uint32_t c_solve_cuda(uint8_t *input, uint8_t *target, uint8_t *nonce, uint32_t gpuid) {
		uint32_t s0,s1,s2,s3,s4,s5,s6,s7,s8,s9,s10,s11,s12,s13,s14,s15;
		uint32_t r0, r1, r2, r3, tmp;
		while(!gpu_divices[gpuid]) {
			gpu_divices[gpuid] = New_GPU_DEVICE();
		}

		uint32_t ret;

		// absorbing
		absorbing(s0, input, 0); absorbing(s1, input, 1);
		absorbing(s2, input, 2); absorbing(s3, input, 3);
		absorbing(s4, input, 4); absorbing(s5, input, 5);
		absorbing(s6, input, 6); absorbing(s7, input, 7);
		s8 = s9 = s10 = s11 = s12 = s13 = s14 = s15 = 0;
		EaglesongPermutation(cpu_injection_constants);

		RAND_bytes((uint8_t*) &r0, 4);
		RAND_bytes((uint8_t*) &r1, 4);
		RAND_bytes((uint8_t*) &r2, 4);
		RAND_bytes((uint8_t*) &r3, 4);

		gpu_divices[gpuid]->state[0] = s0 ^ r0;
		gpu_divices[gpuid]->state[1] = s1 ^ r1;
		gpu_divices[gpuid]->state[2] = s2 ^ r2;
		gpu_divices[gpuid]->state[3] = s3 ^ r3;
		gpu_divices[gpuid]->state[4] = s4 ^ DELIMITER;

		gpu_divices[gpuid]->state[5] = s5; gpu_divices[gpuid]->state[6] = s6;
		gpu_divices[gpuid]->state[7] = s7; gpu_divices[gpuid]->state[8] = s8;
		gpu_divices[gpuid]->state[9] = s9; gpu_divices[gpuid]->state[10] = s10;
		gpu_divices[gpuid]->state[11] = s11; gpu_divices[gpuid]->state[12] = s12;
		gpu_divices[gpuid]->state[13] = s13; gpu_divices[gpuid]->state[14] = s14;
		gpu_divices[gpuid]->state[15] = s15;


		gpu_divices[gpuid]->target = target;
		gpu_divices[gpuid]->nonce_id = 0;

		hipSetDevice(gpuid);
		ret = gpu_hash(gpuid);

		if(gpu_divices[gpuid]->nonce_id) {
			((uint32_t*)nonce)[0] = htobe32(r0^(gpu_divices[gpuid]->nonce_id));
			((uint32_t*)nonce)[1] = htobe32(r1);
			((uint32_t*)nonce)[2] = htobe32(r2);
			((uint32_t*)nonce)[3] = htobe32(r3);
		}

		return ret;
	}
}