#include "hip/hip_runtime.h"
/// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stddef.h>
#include "portable_endian.h"
#include "eaglesong.h"

#define INPUT_LEN (32)
#define N ((INPUT_LEN+8+1)+3) >> 2
#define M (INPUT_LEN >> 2)
#define OUTPUT_LEN 32
#define THREADS_PER_BLOCK  (512)
#define MAX_HASH_NUM (1<<30)
#define MAX_GPU_NUM (1024)
#define HASH_NUM (1<<27)

#define DELIMITER (0x06)

#define ROUND (43)


#define ROL32(a,b) (((a)<<(b))|((a)>>(32-(b))))
#define ROL_ADD(a,b) a += b; a = ROL32(a, 8); b = ROL32(b, 24) + a;

#define EaglesongPermutation() \
{ \
	for(int i = 0, j=0; i < ROUND ; ++i, j+=16) { \
		tmp = s0 ^ s4 ^ s12 ^ s15; s0 = tmp^s5 ^ s6 ^ s7; s1 = tmp^s1 ^ s8 ^ s13; \
		tmp = s1 ^ s2 ^ s6 ^ s14; s2 = tmp^s7 ^ s8 ^ s9; s3 = tmp^s3 ^ s10 ^ s15; \
		tmp = s0 ^ s3 ^ s4 ^ s8; s4 = tmp^s9 ^ s10 ^ s11; s5 = tmp^s1 ^ s5 ^ s12; \
		tmp = s2 ^ s5 ^ s6 ^ s10; s6 = tmp^s11 ^ s12 ^ s13; s7 = tmp^s3 ^ s7 ^ s14; \
		tmp = s4 ^ s7 ^ s8 ^ s12; s8 = tmp^s13 ^ s14 ^ s15; s9 = tmp^s0 ^ s5 ^ s9; \
		tmp = s6 ^ s9 ^ s10 ^ s14; s10 = tmp^s0 ^ s1 ^ s15; s11 = tmp^s2 ^ s7 ^ s11; \
		tmp = s0 ^ s8 ^ s11 ^ s12; s12 = tmp^s1 ^ s2 ^ s3; s13 = tmp^s4 ^ s9 ^ s13; \
		tmp = s3 ^ s5 ^ s13 ^ s14; s14 = tmp^s2 ^ s4 ^ s10; s15 = tmp^s0 ^ s1 ^ s6 ^ s7 ^ s8 ^ s9 ^ s15; \
		s0 ^= ROL32(s0, 2) ^ ROL32(s0, 4) ^ gpu_injection_constants[(j ^ 0)];                    \
		s1 ^= ROL32(s1, 13) ^ ROL32(s1, 22) ^ gpu_injection_constants[(j ^ 1)];                  \
		ROL_ADD(s0, s1);                                                                      \
		s2 ^= ROL32(s2, 4) ^ ROL32(s2, 19) ^ gpu_injection_constants[(j ^ 2)];                   \
		s3 ^= ROL32(s3, 3) ^ ROL32(s3, 14) ^ gpu_injection_constants[(j ^ 3)];                   \
		ROL_ADD(s2, s3);                                                                      \
		s4 ^= ROL32(s4, 27) ^ ROL32(s4, 31) ^ gpu_injection_constants[(j ^ 4)];                  \
		s5 ^= ROL32(s5, 3) ^ ROL32(s5, 8) ^ gpu_injection_constants[(j ^ 5)];                    \
		ROL_ADD(s4, s5);                                                                      \
		s6 ^= ROL32(s6, 17) ^ ROL32(s6, 26) ^ gpu_injection_constants[(j ^ 6)];                  \
		s7 ^= ROL32(s7, 3) ^ ROL32(s7, 12) ^ gpu_injection_constants[(j ^ 7)];                   \
		ROL_ADD(s6, s7);                                                                      \
		s8 ^= ROL32(s8, 18) ^ ROL32(s8, 22) ^ gpu_injection_constants[(j ^ 8)];                  \
		s9 ^= ROL32(s9, 12) ^ ROL32(s9, 18) ^ gpu_injection_constants[(j ^ 9)];                  \
		ROL_ADD(s8, s9);                                                                      \
		s10 ^= ROL32(s10, 4) ^ ROL32(s10, 7) ^ gpu_injection_constants[(j ^ 10)];                 \
		s11 ^= ROL32(s11, 4) ^ ROL32(s11, 31) ^ gpu_injection_constants[(j ^ 11)];                \
		ROL_ADD(s10, s11);                                                                    \
		s12 ^= ROL32(s12, 12) ^ ROL32(s12, 27) ^ gpu_injection_constants[(j ^ 12)];               \
		s13 ^= ROL32(s13, 7) ^ ROL32(s13, 17) ^ gpu_injection_constants[(j ^ 13)];                \
		ROL_ADD(s12, s13);                                                                    \
		s14 ^= ROL32(s14, 7) ^ ROL32(s14, 8) ^ gpu_injection_constants[(j ^ 14)];                 \
		s15 ^= ROL32(s15, 1) ^ ROL32(s15, 13) ^ gpu_injection_constants[(j ^ 15)];                \
		ROL_ADD(s14, s15); \
	} \
}

__constant__ uint32_t gpu_injection_constants[688] = INJECT_MAT;

#define squeeze(s, k) {\
    ((uint32_t *)output)[k] = (s); \
}


struct GPU_DEVICE
{
	uint32_t   state[N];
	uint32_t   nonce_id;
	uint8_t    *target;
	uint32_t    *g_state;
	uint8_t    *g_target;
	uint32_t   *g_nonce_id;
};

GPU_DEVICE *gpu_divices[MAX_GPU_NUM] = {NULL};
uint32_t gpu_divices_cnt = 0;


__global__ void g_eaglesong(uint32_t *state, uint8_t* target, uint32_t *nonce_id)
{
	uint32_t global_id = blockDim.x * blockIdx.x + threadIdx.x;

	uint32_t id = global_id % THREADS_PER_BLOCK;
	uint32_t tmp;
	uint32_t s0, s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, s11, s12, s13, s14, s15;
	uint8_t output[OUTPUT_LEN];

	__shared__ uint32_t shared_state[N];
	__shared__ uint8_t shared_target[OUTPUT_LEN];

	if (id < N) shared_state[id] = state[id]; 
	if (id < OUTPUT_LEN) shared_target[id] = target[id];
	__syncthreads();

	s0 = shared_state[0] ^ (global_id+1);
	s1 = shared_state[1]; s2 = shared_state[2]; s3 = shared_state[3];
	s4 = shared_state[4]; s5 = shared_state[5]; s6 = shared_state[6]; s7 = shared_state[7];
	s8 = s9 = s10 = s11 = s12 = s13 = s14 = s15 = 0;
	
	EaglesongPermutation();
	
	s0 ^= shared_state[8]; s1 ^= shared_state[9]; s2 ^= shared_state[10];
	
	EaglesongPermutation();

	squeeze(s0, 0); squeeze(s1, 1); squeeze(s2, 2); squeeze(s3, 3);
	squeeze(s4, 4); squeeze(s5, 5); squeeze(s6, 6); squeeze(s7, 7);

	for(int k=0; k<32; ++k) {
		if(output[k] < shared_target[k]) {
			atomicExch(nonce_id, global_id+1);
		} else if(output[k] > shared_target[k]) {
			break;
		}
	}
}

int gpu_hash(uint32_t gpuid)
{
	if (HASH_NUM > MAX_HASH_NUM) {
		printf("HASH_NUM out of bound!!!\n");
		return 0;
	}

	if (gpu_divices[gpuid]->g_state == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_state, sizeof(gpu_divices[gpuid]->state)) != hipSuccess) {
			printf("E01: cuda alloc memory error for state\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->g_nonce_id == NULL) 
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_nonce_id, sizeof(gpu_divices[gpuid]->nonce_id)) != hipSuccess) {
			printf("E02: cuda alloc memory error for nonce\n");
			return 0;
		}
	}

	if (gpu_divices[gpuid]->g_target == NULL)
	{
		if (hipMalloc((void **)&gpu_divices[gpuid]->g_target, OUTPUT_LEN) != hipSuccess) {
			printf("E03: cuda alloc memory error for target\n");
			return 0;
		}
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_state, gpu_divices[gpuid]->state, sizeof(gpu_divices[gpuid]->state), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E04: copy memory error for state\n");
		return 0;
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_target, gpu_divices[gpuid]->target, OUTPUT_LEN, hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E05: copy memory error for target\n");
		return 0;
	}

	if (hipMemcpy(gpu_divices[gpuid]->g_nonce_id, &(gpu_divices[gpuid]->nonce_id), sizeof(gpu_divices[gpuid]->nonce_id), hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("E06: copy memory error for nonce\n");
		return 0;
	}

	g_eaglesong << <HASH_NUM / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(gpu_divices[gpuid]->g_state, gpu_divices[gpuid]->g_target, gpu_divices[gpuid]->g_nonce_id);
	hipDeviceSynchronize();

	if (hipMemcpy(&(gpu_divices[gpuid]->nonce_id), gpu_divices[gpuid]->g_nonce_id, sizeof(gpu_divices[gpuid]->nonce_id), hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("E07: copy memory error for g_nonce_id\n");
		return 0;
	}

	return HASH_NUM;
}

GPU_DEVICE* New_GPU_DEVICE()
{
	GPU_DEVICE* p = NULL;
	p = (GPU_DEVICE*)malloc(sizeof(GPU_DEVICE));
	if (p != NULL)
	{
		p->g_target = NULL;
		p->g_nonce_id = NULL;
		p->g_state = NULL;
		p->g_target = NULL;
	} else {
		printf("E08: alloc memory error!\n");
	}
	return p;
}

void RESET_GPU_DEVICE(uint32_t gpuid)
{
	
	memset(gpu_divices[gpuid]->state, 0, sizeof(gpu_divices[gpuid]->state));
	gpu_divices[gpuid]->nonce_id = 0;

	hipFree(gpu_divices[gpuid]->g_nonce_id);
	hipFree(gpu_divices[gpuid]->g_state);
	hipFree(gpu_divices[gpuid]->g_target);
	
	gpu_divices[gpuid]->target = NULL;
	gpu_divices[gpuid]->g_nonce_id = NULL;
	gpu_divices[gpuid]->g_state = NULL;
	gpu_divices[gpuid]->g_target = NULL;
}

void GPU_Count()
{
	int num;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&num);
	printf("deviceCount := %d\n", num);
	gpu_divices_cnt = 0;
	for (int i = 0; i<num; i++)
	{

		hipGetDeviceProperties(&prop, i);
		printf("name:%s\n", prop.name);
		printf("totalGlobalMem:%lu GB\n", prop.totalGlobalMem / 1024 / 1024 / 1024);
		printf("multiProcessorCount:%d\n", prop.multiProcessorCount);
		printf("maxThreadsPerBlock:%d\n", prop.maxThreadsPerBlock);
		printf("sharedMemPerBlock:%lu KB\n", prop.sharedMemPerBlock/1024);
		printf("major:%d,minor:%d\n", prop.major, prop.minor);
		gpu_divices_cnt++;
	}
	if (gpu_divices_cnt > MAX_GPU_NUM)gpu_divices_cnt = MAX_GPU_NUM;
}

extern "C" {
	uint32_t c_solve_gpu(uint8_t *input, uint8_t *target, uint64_t *nonce, uint32_t gpuid) {
		while(!gpu_divices[gpuid]) {
			gpu_divices[gpuid] = New_GPU_DEVICE();
		}

		uint32_t ret;
		RAND_bytes((uint8_t*) &(gpu_divices[gpuid]->state[0]), 4);
		RAND_bytes((uint8_t*) &(gpu_divices[gpuid]->state[1]), 4);
		
		// absorbing
		for(int j = 0, k=0; j <= M; ++j) {
			uint32_t sum = 0;
			for(int v=0; v < 4; ++v) {
				if(k < INPUT_LEN) {
					sum = (sum << 8) ^ input[k];
				} else if(k == INPUT_LEN) {
					sum = (sum << 8) ^ DELIMITER;
				}
				++k;
			}
			gpu_divices[gpuid]->state[j+2] = sum;
		}
		gpu_divices[gpuid]->target = target;
		gpu_divices[gpuid]->nonce_id = 0;

		ret = gpu_hash(gpuid);

		if(gpu_divices[gpuid]->nonce_id) {
			*nonce = le32toh(htobe32(gpu_divices[gpuid]->state[1]));
			*nonce = (*nonce << 32) ^ le32toh(htobe32(((gpu_divices[gpuid]->state[0])^(gpu_divices[gpuid]->nonce_id))));
		}

		return ret;
	}
}